#include "hip/hip_runtime.h"
#include "objDet.h"

/****************************************************************************************************************************************************
 * This function runs the detection of objects with DetectNet.
 * Detectnet expects the input image as a float4 array in RGBAf format.
 * The OpenCV Mat image is in uint8 format. It can be easily converted to uchar3.
 * uchar3 and float4 are special types for DetectNet.
 * The image is copied to the GPU memory and converted to uchar3 during this action.
 * It can then be converted to float4 with the command cudaRGB8ToRGBA32().
 * Once the conversion is done, the detection process is started and the detected objects are returned.
 * If the application is from the Steereocam, the input image needs to be converted from GRAY to BGR first.
 * 
 * Input parameters:
 *  - int *numDetections: A pointer to a integer variable in the calling function.
 *  - detectNet *net: A pointer to the DetectNet object of the calling function.
 *  - cv::Mat *image: This is the input image in OpenCV format.
 *  - const uint32_t overlayFlags: These are the flags that define what DetectNet should return 
 *          (i.e. bounding boxes, labels, confidence)
 *  - uchar3* imgBufferRGB: The buffer for the input image for the final conversion. The data of the input image is copied to this with hipMemcpy2D.
 *  - float4* imgBufferRGBAf: The buffer for the input image that is used by detectnet. imgBufferRGB is converted to float4 with cudaRGB8ToRGBA32.
 * 
 * Return values:
 *  - int *numDetections: After the detection process is finished, it holds the number of detected objects.
 *  - detectNet::Detection *detections: This is a structure that holds the detected objects 
 *          including the elements defined in overlayFlags * 
 * **************************************************************************************************************************************************/
detectNet::Detection* objDet_run(int *numDetections, detectNet* net, cv::Mat *image, const uint32_t overlayFlags, uchar3* imgBufferRGB, float4* imgBufferRGBAf)
{
#ifdef STEEREOCAM
    cv::cvtColor(*image, *image, cv::COLOR_GRAY2BGR);
#endif
    
    detectNet::Detection* detections = NULL;
    
/********************************************************************************
 * Here the OpenCV Mat is copied to the buffer in the GPU memory and converted to float4 format
 * ******************************************************************************/
    hipMemcpy2D((void*)imgBufferRGB, image->cols*sizeof(uchar3), (void*)image->data, image->step, image->cols*sizeof(uchar3), image->rows, hipMemcpyHostToDevice);
    cudaRGB8ToRGBA32(imgBufferRGB, imgBufferRGBAf, image->cols, image->rows);
    
/***************************************
 * Here the detection of objects is done
 * *************************************/
    *numDetections = net->Detect((float*)imgBufferRGBAf, (uint32_t)image->cols, (uint32_t)image->rows, &detections, overlayFlags);
    
/********************************
 * Returning the detected objects
 * ******************************/
    return detections;
}

/*******************************************************************************************************************************************
 * This function calculates the 3D coordinates of the detected objects.
 * In case the coordinates should be written to a shared memory, they are copied to an object of type XYZ which is a pointer to the shared memory.
 * First, the center of the detected objects is calculated.
 * 
 * Realsense-application:
 * Once this is done, the function gets the depth value for this specific pixel in the realsense-application.
 * If post-processing filters are activated for the depth map, the pixel needs to be adjusted according to the defined decimation magnitude.
 * If the depth value is greater than 0, the 3D coordinates can be calculated with the help  of the function rs2_deproject_pixel_to_point().
 * rs2_deproject_pixel_to_point() calculates the 3D coordinates with the camera intrinsics of the color camera. 
 * Therefore it needs the pixel of the color image as input.
 * This function returns the X, Y and Z coordinate in a float array.
 * 
 * Steereocam-application:
 * After calculating the center of the detected objects, the coordinates can directly be obtained from cv::Mat xyz.
 * This cv::Mat holds all 3D values of the image and needs to be calculated before this function using cv::reprojectImageTo3D.
 * 
 * The next step is to sort the detected objects by their distance (ascending).
 * At the end of the function, the coordinates can be printed on the color image, logged to console, 
 * or copied to the XYZ object into the shared memory. This depends on the defines #SHOW_IM, #LOGOBJECTS and #USESHM.
 * 
 * Input parameters:
 *  - detectNet::Detection* detections: Structure that holds the detected objects.
 *  - const int numDetections: Number of objects that were detected.
 *  - detectNet* net: A pointer to the DetectNet object of the calling function. This is needed for getting the object class (i.e. TV).
 *  - cv::Mat *image: This is the color image where the objects were detected.
 *  - XYZ *coordinates: A pointer to an object of type XYZ that is used as shared memory.
 *  - sem_t* semptr: A pointer to the semaphore that is used for synchronization of the memory access of the shared memory.
 * 
 * Realsense-specific input parameters:
 *  - rs2::depth_frame *depth: A pointer to the depth frame.
 *  - const rs2_intrinsics color_intrin: This are the intrinsic parameters of the color camera.
 * Steereocam-specific input parameters:
 *  - cv::Mat *xyz: This cv::Mat holds the 3D values of every pixel. This is the output of reprojectImageTo3D.
 * 
 * Return values:
 *  - cv::Mat *image: The color image with bounding boxes, labels and coordinates of the detected objects.
 *  - XYZ *coordinates: The function copies the detected objects and coordinates to this structure that is used as shared memory. 
 * *****************************************************************************************************************************************/
#ifdef USESHM

#ifdef STEEREOCAM
void objDet_calc_coordinates(detectNet::Detection* detections, const int numDetections, detectNet* net, cv::Mat *xyz, cv::Mat *image, XYZ *coordinates, sem_t* semptr)
#else
void objDet_calc_coordinates(detectNet::Detection* detections, const int numDetections, detectNet* net, rs2::depth_frame *depth, const rs2_intrinsics color_intrin, cv::Mat *image, XYZ *coordinates, sem_t* semptr)
#endif

#else

#ifdef STEEREOCAM
void objDet_calc_coordinates(detectNet::Detection* detections, const int numDetections, detectNet* net, cv::Mat *xyz, cv::Mat *image)
#else
void objDet_calc_coordinates(detectNet::Detection* detections, const int numDetections, detectNet* net, rs2::depth_frame *depth, const rs2_intrinsics color_intrin, cv::Mat *image)
#endif

#endif
{
    float pixel_point[2], depth_point[3];
    float calc_dist;
#if defined(SHOW_IM) || defined(USESHM)
    int numIters = 0;
#endif
    cv::Scalar color_bb = cv::Scalar(255, 10, 255); // magenta
    cv::Scalar color_text = cv::Scalar(0, 200, 200); // yellow
    tempObjects tempCoords[numDetections];

#ifdef REALSENSE
    float depth_val;
    #ifdef FILTER
        float pixel_pointFilt[2];
    #endif
#endif
        
#ifdef STEEREOCAM
    cv::Point3f depth_steereo;
#endif
    
/**************************************************************************************************************************
 * This loop loops through the detected objects and calculates the depth and 3D coordinates of the center of the objects.
 * ************************************************************************************************************************/
    for(int n=0; n < numDetections; n++)
    {
/************************************************************************************
 * Here the detected objects are copied to a temporary structure of type tempObjects.
 * This structure is used for sorting the objects by their distance.
 * **********************************************************************************/
        tempCoords[n].detection = &detections[n];
        
/**********************************************************************************************************
 * Here the pixel of the center of the detected objects is calculated.
 * In case #COPTER is defined, it means that the image was rotated by 180°.
 * As only the color image was rotated by the main function, the pixel needs to be adjusted for the depth map.
 * ********************************************************************************************************/
#ifdef COPTER                
        pixel_point[0] = WIDTH - detections[n].Left - detections[n].Width()*0.5;
        pixel_point[1] = HEIGHT - detections[n].Top - detections[n].Height()*0.5;
#else
        pixel_point[0] = detections[n].Left + detections[n].Width()*0.5;
        pixel_point[1] = detections[n].Top + detections[n].Height()*0.5;
#endif   

        
#ifdef REALSENSE
/****************************************************************************************************************
 * If the post-processing filters of the depth map are active, the depth image is reduced by the factor DEC_MAGN.
 * Therefore the pixel needs to be adjusted for getting the correct depth value.
 * If they are not active, the above calculated pixel can be used.
 * Also, the distance of the pixel in meters is obtained. This is needed for calculating the 3D coordinates later.
 * **************************************************************************************************************/
    #if defined(FILTER)              
            pixel_pointFilt[0] = pixel_point[0]/DEC_MAGN;
            pixel_pointFilt[1] = pixel_point[1]/DEC_MAGN;         
            depth_val = depth->get_distance((int)pixel_pointFilt[0], (int)pixel_pointFilt[1]);
    #else
            depth_val = depth->get_distance((int)pixel_point[0], (int)pixel_point[1]);
    #endif
#else
/**********************************************************************************
 * For the Steereocam the 3D coordinates can be directly obtained from cv::Mat xyz.
 * ********************************************************************************/
            depth_steereo = xyz->at<cv::Point3f>(pixel_point[1], pixel_point[0]);
            depth_point[0] = depth_steereo.x;
            depth_point[1] = depth_steereo.y;
            depth_point[2] = depth_steereo.z;
#endif

#ifdef REALSENSE
/****************************************************************************************************************
 * If the depth value is greater than 0, the 3D coordinates of the corresponding pixel are calculated.
 * The real distance is then calculated with the formula Dist = SQRT(X*X + Y*Y + Z*Z).
 * As the realsense uses an other coordinate system where the positive Y-axis points down 
 * (see https://dev.intelrealsense.com/docs/projection-in-intel-realsense-sdk-20),
 * the value of the Y-coordinate is multiplied with -1 to convert it to the coordinate system used by the copter.
 * In case the #COPTER is defined, the image is rotated by 180° what means that the Y-coordinate is fine, 
 * but the X-coordinate needs to be multiplied with -1.
 * 
 * If the depth value is 0 or lower, it means that the depth value could not be obtained by the camera.
 * In this case, default values (defined in "global_defines.h") are assigned to the coordinates.
 * **************************************************************************************************************/
        if(depth_val > 0)  
        {
/*************************************************************************************************
 * In rs2_deproject_pixel_to_point the 3D coordinates of a pixel in the color image are calculated 
 * by using the camera intrinsics of the color camera and the obtained depth value.
 * ***********************************************************************************************/
            rs2_deproject_pixel_to_point(depth_point, &color_intrin, pixel_point, depth_val);
#endif
 
/**************************************************************************************************************************
 * Here the real distance to the object is calculated with the 3D coordinates and the formula Dist = SQRT(X*X + Y*Y + Z*Z).
 * ************************************************************************************************************************/
            calc_dist = sqrt(depth_point[0] * depth_point[0] + depth_point[1] * depth_point[1] + depth_point[2] * depth_point[2]);
    
/*****************************************************************
 * Here the X or Y axis are multiplied with -1 as described above.
 * ***************************************************************/
#ifdef COPTER
            depth_point[0] = depth_point[0]*(-1);
#else
            depth_point[1] = depth_point[1]*(-1);
#endif
            
#ifdef REALSENSE
        }
#endif

/*************************************************************************************************
 * If the depth value could not be calculated, default values will be assigned to the coordinates.
 * In the realsense calculations, the depth value is 0 in case it could not be calculated.
 * In the steerecam calculations it is infinite.
 * ***********************************************************************************************/
#ifdef STEEREOCAM
        if(isinf(calc_dist)) 
#else
        else
#endif
        {
/**********************************************************
 * Here the default values are assigned as described above.
 * ********************************************************/
            depth_point[0] = DEFAULT_X_COORD;
            depth_point[1] = DEFAULT_Y_COORD;
            depth_point[2] = DEFAULT_Z_COORD;
            calc_dist = DEFAULT_DISTANCE;
        }
        
/****************************************************************************************************
 * Here the 3D coordinates are copied to the temporary structure for sorting the objects by distance.
 * **************************************************************************************************/
        tempCoords[n].depth_point[0] = depth_point[0];
        tempCoords[n].depth_point[1] = depth_point[1];
        tempCoords[n].depth_point[2] = depth_point[2];
        tempCoords[n].calc_dist = calc_dist;
    }

/*********************************************************************
 * In this part, the objects are sorted by their distance (ascending).
 * *******************************************************************/
	for(int i=0; i < numDetections-1; i++)
	{
		for(int j=0; j < numDetections-i-1; j++)
		{
			if(tempCoords[j].calc_dist > tempCoords[j+1].calc_dist)
			{
				const tempObjects det = tempCoords[j];
				tempCoords[j] = tempCoords[j+1];
				tempCoords[j+1] = det;
			}
		}
	}
    
#ifdef LOGOBJECTS
    std::cout << "Found objects:" << std::endl;
#endif    
    
/**********************************************************************************************************
 * If #SHOW_IM is defined, the image is manipulated and the bounding boxes and coordinates are drawn to it.
 * If #USESHM is defined, the coordinates are copied to the XYZ object of the calling function that is used as shared memory..
 * ********************************************************************************************************/
#if defined(SHOW_IM) || defined(USESHM)
/*********************************************************************
 * With #MAXOBJECTS the maximum number of show objects can be limited.
 * *******************************************************************/
    if(numDetections > MAXOBJECTS)
    {
        numIters = MAXOBJECTS;
    }
    else
    {
        numIters = numDetections;
    }
#endif

#ifdef USESHM
	/***************************************************************************************
	 * If the number of found objects is smaller than #MAXOBJECTS, 
	 * the other elements of the shared memory structure get default values.
	 * The semaphore is locked until all objects are written to the shared memory structure.
	 * *************************************************************************************/
	if(sem_wait(semptr) == 0)
    {
	/**********************************************************************************************
	 * In this part, the 3D coordinates and the distance are copied to the shared memory structure.
	 * ********************************************************************************************/
		for(int n=0; n < numIters; n++)
		{
            std::string tempStr = net->GetClassDesc(tempCoords[n].detection->ClassID);
            strcpy(coordinates[n].objType, tempStr.c_str());
            coordinates[n].x_coord = tempCoords[n].depth_point[0];
            coordinates[n].y_coord = tempCoords[n].depth_point[1];
            coordinates[n].z_coord = tempCoords[n].depth_point[2];
            coordinates[n].distance = tempCoords[n].calc_dist;
		}
        if(numIters < MAXOBJECTS)
        {
            for(int n=numIters; n < MAXOBJECTS; n++)
            {
                std::string tempStr = "None";
                strcpy(coordinates[n].objType, tempStr.c_str());
                coordinates[n].x_coord = DEFAULT_X_COORD;
                coordinates[n].y_coord = DEFAULT_Y_COORD;
                coordinates[n].z_coord = DEFAULT_Z_COORD;
                coordinates[n].distance = DEFAULT_DISTANCE;
            }
        }
		/**************************
		 * Releasing the semaphore.
		 * ************************/
		sem_post(semptr);
	}
#endif
#ifdef SHOW_IM 
        for(int n=0; n < numIters; n++)
        {
/*************************************************************************************
 * Here the bounding boxes are drawn and the type of object is printed on the image.
 * ***********************************************************************************/
            cv::Point p1(tempCoords[n].detection->Left + tempCoords[n].detection->Width(), tempCoords[n].detection->Top + tempCoords[n].detection->Height());
            cv::Point p2(tempCoords[n].detection->Left, tempCoords[n].detection->Top);
            cv::putText(*image, net->GetClassDesc(tempCoords[n].detection->ClassID), cv::Point(tempCoords[n].detection->Left, (tempCoords[n].detection->Top + 25)), cv::FONT_HERSHEY_SIMPLEX, 0.75, color_text, 2);
            cv::rectangle(*image, p1, p2, color_bb, 2, 8, 0);

/*****************************************************************************
 * Here the distance and the 3D coordinates are printed on the image.
 * Also, the center of the object is marked with a rectangle of 10x10 pixels.
 * ***************************************************************************/
            cv::Point p3(tempCoords[n].detection->Left + (tempCoords[n].detection->Width() / 2) + 10, tempCoords[n].detection->Top + (tempCoords[n].detection->Height() / 2) + 10);
            cv::Point p4(tempCoords[n].detection->Left + (tempCoords[n].detection->Width() / 2) - 10, tempCoords[n].detection->Top + (tempCoords[n].detection->Height() / 2) - 10);
            cv::rectangle(*image, p3, p4, color_bb, 2, 8, 0);
            cv::putText(*image, "Distance: " + std::to_string(tempCoords[n].calc_dist) + "m", cv::Point(tempCoords[n].detection->Left, (tempCoords[n].detection->Top + 50)), cv::FONT_HERSHEY_SIMPLEX, 0.75, color_text, 2);
            cv::putText(*image, "X: " + std::to_string(tempCoords[n].depth_point[0]) + "m", cv::Point(tempCoords[n].detection->Left, (tempCoords[n].detection->Top + 75)), cv::FONT_HERSHEY_SIMPLEX, 0.75, color_text, 2);
            cv::putText(*image, "Y: " + std::to_string(tempCoords[n].depth_point[1]) + "m", cv::Point(tempCoords[n].detection->Left, (tempCoords[n].detection->Top + 100)), cv::FONT_HERSHEY_SIMPLEX, 0.75, color_text, 2);
            cv::putText(*image, "Z: " + std::to_string(tempCoords[n].depth_point[2]) + "m", cv::Point(tempCoords[n].detection->Left, (tempCoords[n].detection->Top + 125)), cv::FONT_HERSHEY_SIMPLEX, 0.75, color_text, 2);        
        }
#endif 

/********************************************************************************************
 * If #LOGOBJECTS is defined, the objects and their 3D coordinates are logged to the console.
 * ******************************************************************************************/
#ifdef LOGOBJECTS
    for(int n=0; n < numDetections; n++)
    {
        std::cout << "Object " << n << ": " << net->GetClassDesc(tempCoords[n].detection->ClassID) << ", X=" << \
            tempCoords[n].depth_point[0] << "[m] Y=" << tempCoords[n].depth_point[1] << "[m] Z=" << tempCoords[n].depth_point[2] << \
            "[m] Distance=" << tempCoords[n].calc_dist << "[m]" << std::endl;
    }    
#endif
}
