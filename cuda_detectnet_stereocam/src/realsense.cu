#include "hip/hip_runtime.h"
#include "realsense.h"

// global variable for the handling of the endless loop
int flag;

/*******************************************************************************************************************
 * This is the main function of the realsense application.
 * It installs a signal handler to the signal SIGINT, initializes the camera module and creates a detectnet object.
 * If defined, it also initializes the gpio, clock and shared memory, colorizer for the depth map, 
 * filters, and sets the maximum laser power.
 * 
 * In an endless loop it obtains the color and depth frames from the realsense camera.
 * It then calls the necessary functions to detect objects and calculate its coordinates.
 * If defined, it also logs the runtime information using the clock, 
 * controls the gpios for external runtime analysis with an oscilloscope for example, and shows the depth frame.
 * If #USESHM is defined, the objects are written to the shared memory in the object detection functions, 
 * or, if no object was found some default values are written to the shared memory by the main function.
 * 
 * The main function includes the following options - these are described and can be activated in global_defines.h:
 * #SHOW_IM, #SHOW_DEPTH, #LOGCONSOLE, #LOGOBJECTS, #FILTER, #MAXLASERPWR,
 * #COPTER, #GPIO_RUNTIME, #CLOCK_RUNTIME, #USESHM, #SAVEPIC, #MAXOBJECTS
 * 
 * The application exits if a SIGINT was caught or, in case #SAVEPIC is defined, it ends after 1000 loops.
 ******************************************************************************************************************/
int main(int argc, char** argv)
{
    // variable for the return value of the application
    int retval = EXIT_SUCCESS;
    
    // installing signal handler
    if(signal(SIGINT, sig_handler) == SIG_ERR)
    {
        perror("signal");
        return EXIT_FAILURE;
    }
    
/***********************************************************
 * Variables for object detection.
 * The obtained color image is copied to these image buffers 
 * and converted to the correct format needed by detectnet.
 * *********************************************************/
    uchar3* imgBufferRGB = NULL;
    float4* imgBufferRGBAf = NULL;
    hipMalloc((void**)&imgBufferRGB, WIDTH * sizeof(uchar3) * HEIGHT);
    hipMalloc((void**)&imgBufferRGBAf, WIDTH * sizeof(float4) * HEIGHT);
    
/**********************************************************
 * Variables used for interfacing with the realsense camera
 * ********************************************************/
    rs2::pipeline p;
    rs2::config cfg;
    rs2::pipeline_profile prof;    
    rs2::align align_to_color(RS2_STREAM_COLOR);
    rs2::frameset frames;
#ifdef FILTER
    rs2::decimation_filter dec_filter;
    rs2::threshold_filter thr_filter;
    rs2::spatial_filter spat_filter;
    rs2::temporal_filter temp_filter;
    rs2::disparity_transform depth_to_disparity(true);
    rs2::disparity_transform disparity_to_depth(false);
#endif

/***************************************************************************************************
 * Variables used for interfacing with detectnet
 * cmdLine holds the attributes from calling the application in the console.
 * With the attribute "--network=" the model for the object-detection can be configured.
 * The possible values depend on which models are downloaded with the Jetson-Inference Model-Loader.
 * This application was tested with the networks "mobilenet-v1" and "mobilenet-v2".
 * If no attribute is given to the application, the default model is mobilenet-v2.
 * In overlayFlags it can be configured, what DetectNet should return.
 * box,labels,conf means bounding boxes, labels (which object was detected), and the confidence.
 * *************************************************************************************************/
    detectNet* net;
    commandLine cmdLine(argc, argv, (const char*)NULL);
    const uint32_t overlayFlags = detectNet::OverlayFlagsFromStr("box,labels,conf");
    detectNet::Detection* detections = NULL;
    int numDetections;
    
/**************************************
 * OpenCV Mat object which is used for:
 * - detecting objects
 * - showing the picture on the screen
 * ************************************/
    cv::Mat image;
    
#ifdef CLOCK_RUNTIME
/******************************************************************
 * Variables used for showing the runtime needed by complete cycle, 
 * grabbing the frames, detecting the objects, 
 * calculating and drawing the coordinates to the image 
 * and filtering the depth frame using the clock
 * ****************************************************************/
    timeval totalStartFPS, totalStartFRAME, totalStartOBJDET;
    #ifdef FILTER
    timeval totalStartFILT;
    #endif
#endif

#if defined(USESHM) || defined(GPIO_RUNTIME) || defined(CLOCK_RUNTIME)
/*****************************************************************
 * Variable used for storing the return value of several functions 
 * ***************************************************************/
    int ret;
#endif
    
#ifdef SAVEPIC
/********************************************************************
 * Counter used for limiting the loop when pictures are saved on disk
 * ******************************************************************/
    int counter = 0;
#endif

#ifdef SHOW_DEPTH
/**********************************************************************************************************************
 * Variables and objects used for coloring the depth map for visualization
 * rs2::colorizer can be configured by setting the option RS2_OPTION_COLOR_SCHEME.
 * This option configures the used colormap. Default is Jet Colormap (0.f).
 * All available option values can be found in rs_processing.hpp from Librealsense in "class colorizer : public filter"
 * ********************************************************************************************************************/
    rs2::colorizer color_map;
    color_map.set_option(RS2_OPTION_COLOR_SCHEME, COLORSCHEME);
#endif

#ifdef GPIO_RUNTIME
    #ifdef LOGCONSOLE
    std::cout << "Initializing GPIO..." << std::endl;
    #endif
/********************************************************************
 * Initialization of the GPIOs.
 * The defined pins are exported, their direction is set to Output
 * and their value is set to 0.
 * In case of error, the error message is printed in the subfunctions
 * and the application exits.
 * ******************************************************************/
    ret = gpio_init();
    if(ret < 0)
    {
#if defined(USESHM)
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, -1, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, 1);
#endif
        return EXIT_FAILURE;
    }

#endif

#ifdef LOGCONSOLE
    std::cout << "Initializing Camera..." << std::endl;
#endif
/************************************************************
 * Initialization of the camera.
 * In this function, the depth and color frames are enabled
 * with the defined resolution and a framerate of 30 FPS.
 * The resolution can be configured in "global_defines.h".
 * **********************************************************/
    camera_init(&cfg, &p, &prof);
    
#ifdef LOGCONSOLE
    std::cout << "Loading Camera Intrinsics..." << std::endl;
#endif
/**********************************************************************
 * Loading the camera intrinsics of the color frame.
 * The intrinsics are needed for the calculation of the 3D coordinates.
 * ********************************************************************/
    const rs2_intrinsics color_intrin = get_camera_intrinsics(&frames, &p);

#ifdef FILTER
    #ifdef LOGCONSOLE
    std::cout << "Initializing filters..." << std::endl;
    #endif
/*********************************************************************
 * Initialization of postprocessing filters for the depth frame.
 * The values are defined in camera.h. 
 * The default values are the default values used in realsense-viewer.
 * *******************************************************************/
    filters_init(&dec_filter, &thr_filter, &spat_filter, &temp_filter);
#endif
    
#ifdef MAXLASERPWR
    #ifdef LOGCONSOLE
    std::cout << "Setting Laser Power to maximum value..." << std::endl;
    #endif
/****************************************************************
 * Setting the power of the infrared laser to its maximum value.
 * The default setting in Librealsense is half the power (180mW).
 * This option increases the power consumption, 
 * but it also increases the quality of the depth frame.
 * **************************************************************/
    set_max_laser_pwr(&prof);

#endif 

#ifdef LOGCONSOLE
    std::cout << "Initializing DetectNet..." << std::endl;
#endif
/*******************************************************************
 * Initialization of DetectNet with the given attributes in cmdLine.
 * The attributes for cmdLine are described above.
 * *****************************************************************/
    net = detectNet::Create(cmdLine);
    if(!net)
    {
        std::cerr << "detectNet: failed to load detectNet model" << std::endl;
#if defined(USESHM) && defined(GPIO_RUNTIME)
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, -1, 1);
#elif defined(USESHM) && !defined(GPIO_RUNTIME)
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, -1);
#elif defined(GPIO_RUNTIME) && !defined(USESHM)
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p);
#endif
        return EXIT_FAILURE;
    }
    
#ifdef USESHM
/***********************************************************************************************
 * Here the shared memory is initialized and mapped to the local XYZ variable.
 * Also, the named semaphore is opened.
 * CAUTION: the name of the semaphore needs to be equal to the name in the read_shm application.
 * *********************************************************************************************/
 #ifdef LOGCONSOLE
    std::cout << "Creating shared memory and semaphore..." << std::endl;
#endif
    // check for a valid key
    if(KEY_RS == -1)
    {
        fprintf(stderr, "invalid key\n");
#ifdef GPIO_RUNTIME            
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, -1, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, -1);
#endif
        return EXIT_FAILURE;
    }
    key_t key = KEY_RS;
    int shmid = shmget(key, sizeof(XYZ)*MAXOBJECTS, 0666|IPC_CREAT);
    if(shmid == -1)
    {
        perror("shmget");
#ifdef GPIO_RUNTIME            
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, shmid, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, shmid);
#endif
        return EXIT_FAILURE;
    }
    XYZ *coordinates = (XYZ*) shmat(shmid, (void*)0, 0);
    if(coordinates == (void*)-1)
    {
        perror("shmat");
#ifdef GPIO_RUNTIME            
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, shmid, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, NULL, shmid);
#endif
        return EXIT_FAILURE;
    }
    sem_t* semptr = sem_open("sem_rs", O_CREAT, 0666, 0);
    if(semptr == SEM_FAILED)
    {
        perror("sem_open");
#ifdef GPIO_RUNTIME            
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, coordinates, shmid, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, NULL, coordinates, shmid);
#endif        
        return EXIT_FAILURE;
    }
    if(sem_post(semptr) == -1)
    {
#ifdef GPIO_RUNTIME            
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, semptr, coordinates, shmid, 1);
#else
        cleanup(imgBufferRGB, imgBufferRGBAf, &p, semptr, coordinates, shmid);
#endif        
        return EXIT_FAILURE;
    }

#endif
    
/***************************************************
 * Checks if a SIGINT signal was already received,
 * if not, the camera and detection loop is started.
 * *************************************************/
#ifdef LOGCONSOLE
    if(flag == 0)
    {
        std::cout << "Starting camera and detection loop..." << std::endl;
    }
#endif

/***********************************************************************************************************************************
 * This is an infinite loop that has the following tasks:
 * - Grabbing the color and depth frame
 * - If configured, applying post-processing filters on the depth frame
 * - Detecting objects in the color frame
 * - Calculating the 3D coordinates of the detected objects and sorting them by distance, ascending
 * - If configured, drawing the bounding boxes of the objects to the image and visualizing it
 * - If configured, visualizing the depth frame
 * - If configured, the 3D coordinates are written to a shared memory in the object detection functions,
 *   or, if no object was found some default values are written to the shared memory by the main function. 
 * 
 * The application exits, if CTRL-C is hit in the command window (this raises a SIGINT signal).
 * In case #SHOW_IM or #SHOW_DEPTH is defined, the loop also ends if ESC is hit in the window that shows the image or the depth map.
 * In case #SAVEPIC is defined, the loop ends after 1000 cycles.
 * *********************************************************************************************************************************/
    while(flag == 0)
    { 
        
#ifdef CLOCK_RUNTIME
    #ifdef LOGCONSOLE
        std::cout << "Loading current time for FPS and FRAME..." << std::endl;
    #endif
/*****************************************************************************************
 * gettimeofday loads the current time since the Epoch.
 * The structs hold the time in seconds and microseconds.
 * The values obtained here are used for calculating the time needed for a complete cycle,
 * and for grabbing the frames.
 * ***************************************************************************************/
        ret = gettimeofday(&totalStartFPS, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not get current start FPS time..." << std::endl;
        }
        ret = gettimeofday(&totalStartFRAME, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not get current start FRAME time..." << std::endl;
        }
#endif
        
#ifdef GPIO_RUNTIME
/************************************************************************************************************
 * sets the value of PIN_FPS and PIN_FRAME to 1 to signal the start of a cycle and obtaining a frame via gpio
 * **********************************************************************************************************/
        ret = gpio_set_value(PIN_FPS, 1);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO FPS to 1..." << std::endl;
        }
        ret = gpio_set_value(PIN_FRAME, 1);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO FRAME to 1..." << std::endl;
        }
#endif
        
#ifdef LOGCONSOLE
        std::cout << "Loading current frames..." << std::endl;
#endif
/************************************************************************************
 * In get_frames the color and depth frame are grabbed.
 * The function returns the depth frame and converts the color frame to a OpenCV Mat.
 * The color frame is saved in the Mat "image".
 * **********************************************************************************/
        rs2::depth_frame depth = get_frames(&frames, &p, &align_to_color, &image);

#ifdef GPIO_RUNTIME
/**********************************************************************************
 * sets the value of PIN_FRAME to 0 to signal the end of obtaining a frame via gpio
 * ********************************************************************************/
        ret = gpio_set_value(PIN_FRAME, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO FRAME to 0..." << std::endl;
        }
#endif
        
#ifdef CLOCK_RUNTIME
/******************************************************************
 * Here the current time is loaded and the time difference 
 * to the time value before "get_frames" is calculated and printed.
 * ****************************************************************/
        print_time(&totalStartFRAME, "Time for grabbing the frames: ");        
#endif
        
#ifdef FILTER
/**************************************************************************************************************
 * In this part, the filtering of the depth frame takes place.
 * In case #CLOCK_RUNTIME is defined, also the time is measured how long the filtering process takes. 
 * In case #GPIO_RUNTIME is defined, also the gpio PIN_FILTER is set to 1 and 0 to output the runtime via gpio.
 * ************************************************************************************************************/
    #ifdef CLOCK_RUNTIME
        #ifdef LOGCONSOLE
        std::cout << "Loading current time for FILTER..." << std::endl;
        #endif
        ret = gettimeofday(&totalStartFILT, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not get current start FILTER time..." << std::endl;
        }
    #endif
        
    #ifdef GPIO_RUNTIME
        ret = gpio_set_value(PIN_FILTER, 1);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO FILTER to 1..." << std::endl;
        }
    #endif
        
    #ifdef LOGCONSOLE
        std::cout << "Start filtering the depth map..." << std::endl;
    #endif
        
        rs2::depth_frame filtered = filter_depth_frame(&depth, &dec_filter, &thr_filter, &depth_to_disparity, &spat_filter, &temp_filter, &disparity_to_depth, &align_to_color);   

    #ifdef GPIO_RUNTIME
        ret = gpio_set_value(PIN_FILTER, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO FILTER to 0..." << std::endl;
        }
    #endif
        
    #ifdef CLOCK_RUNTIME
        print_time(&totalStartFILT, "Time for filtering the depth frame: ");
    #endif
#endif
    
#ifdef COPTER
/*****************************************************************************************
 * On the copter, the camera is rotated by 180° and so the image also needs to be rotated.
 * This is essential for the visualization, but also for the object detection.
 * ***************************************************************************************/
    #ifdef LOGCONSOLE
        std::cout << "Rotating image for the copter..." << std::endl;
    #endif
        rotate(image, image, 1);

#endif
        
/****************************************************************************************************************
 * Here the function for detecting objects in the color image is called.
 * It converts the OpenCV Mat to a float4 array and loads it to the GPU,
 * as it is needed in this format by DetectNet.
 * Then the detection with DetectNet is done and the found objects are returned with the configured overlayFlags. 
 * In case #CLOCK_RUNTIME is defined, also the time is measured how long the object detection takes. 
 * In case #GPIO_RUNTIME is defined, also the gpio PIN_OBJDET is set to 1 and 0 to output the runtime via gpio.
 * **************************************************************************************************************/
#ifdef CLOCK_RUNTIME
    #ifdef LOGCONSOLE
        std::cout << "Loading current time for OBJDET..." << std::endl;
    #endif
        ret = gettimeofday(&totalStartOBJDET, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not get current start OBJDET time..." << std::endl;
        }
#endif
        
#ifdef GPIO_RUNTIME
        ret = gpio_set_value(PIN_OBJDET, 1);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO OBJDET to 1..." << std::endl;
        }
#endif
        
#ifdef LOGCONSOLE
        std::cout << "Starting object detection..." << std::endl;
#endif
        // running object detection
        detections = objDet_run(&numDetections, net, &image, overlayFlags, imgBufferRGB, imgBufferRGBAf);

#ifdef GPIO_RUNTIME
        ret = gpio_set_value(PIN_OBJDET, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO OBJDET to 0..." << std::endl;
        }
#endif
        
#ifdef CLOCK_RUNTIME
/******************************************************************
 * Here the current time is loaded and the time difference 
 * to the time value before "objDet_run" is calculated and printed.
 * ****************************************************************/
        print_time(&totalStartOBJDET, "Time for object detection: ");
#endif
        
/********************************************************************************************************
 * If DetectNet found 1 or more objects in the color image,
 * the depth and the 3D coordinates are calculated in objDet_calc_coordinates.
 * If #SHOW_IM is defined, the bounding boxes and the coordinates are drawn to the color image.
 * In case of active filters, the filtered depth frame is used for getting the depth.
 * In case the coordinates should be written to a shared memory, this is done in objDet_calc_coordinates.
 * If no object was found, some default values are written to the shared memory by the main function.
 * ******************************************************************************************************/
        if(numDetections > 0)
        {
#ifdef LOGCONSOLE
            std::cout << "Starting calculation of coordinates..." << std::endl;
#endif
            
#ifdef USESHM
    #ifdef FILTER
            objDet_calc_coordinates(detections, numDetections, net, &filtered, color_intrin, &image, coordinates, semptr);
    #else
            objDet_calc_coordinates(detections, numDetections, net, &depth, color_intrin, &image, coordinates, semptr);
    #endif
#else
    #ifdef FILTER
            objDet_calc_coordinates(detections, numDetections, net, &filtered, color_intrin, &image);
    #else
            objDet_calc_coordinates(detections, numDetections, net, &depth, color_intrin, &image);
    #endif        
#endif
        }

#ifdef USESHM
        else
        {
			if(sem_wait(semptr) == 0)
			{
				for(int n=0; n < MAXOBJECTS; n++)
				{
					std::string tempStr = "None";
					strcpy(coordinates[n].objType, tempStr.c_str());
					coordinates[n].x_coord = DEFAULT_X_COORD;
					coordinates[n].y_coord = DEFAULT_Y_COORD;
					coordinates[n].z_coord = DEFAULT_Z_COORD;
					coordinates[n].distance = DEFAULT_DISTANCE;
				}
				sem_post(semptr);
			}
        }
#endif

#ifdef SHOW_IM
/********************************************************
 * cv::imshow creates a window and visualizes color image
 * ******************************************************/
    #ifdef LOGCONSOLE
        std::cout << "Showing color image..." << std::endl;
    #endif
        cv::imshow("RealSense Image", image);
#endif
        
#ifdef SHOW_DEPTH
/***************************************************************************
 * In this part, the depth frames are colored using a rs2::colorizer object.
 * To visualize it, the frames need to be converted to an OpenCV Mat.
 * Once done, they are visualized with cv::imshow.
 * *************************************************************************/
    #ifdef LOGCONSOLE
        std::cout << "Showing depth map..." << std::endl;
    #endif
        
        show_depth_frame(&color_map, &depth, "");
    
    #ifdef FILTER 
        #ifdef LOGCONSOLE
        std::cout << "Showing filtered depth map..." << std::endl;
        #endif
        
        show_depth_frame(&color_map, &filtered, "filtered");
    #endif    
#endif

#ifdef SAVEPIC
/****************************************************************************
 * In this part, the color image is saved in the defined location.
 * The name of the saved images includes a number that counts up.
 * The location and prefix and the format are defined in global_defines.h.
 * **************************************************************************/
    #ifdef LOGCONSOLE
        std::cout << "Saving color image to disk..." << std::endl;
    #endif
        std::string name = PICLOCATIONANDPREFIX + std::to_string(counter) + PICFORMAT;
        cv::imwrite(name, image);
        counter++;
        if(counter > 999)
            break;
#endif
     
#ifdef GPIO_RUNTIME
/***********************************************************************
 * Sets the value of PIN_FPS to 0 to signal the end of a cycle via gpio.
 * *********************************************************************/
        ret = gpio_set_value(PIN_FPS, 0);
        if(ret < 0)
        {
            // not a critical error, no further action needed.
            std::cerr << "Could not set GPIO FPS to 0..." << std::endl;
        }
#endif
        
#ifdef CLOCK_RUNTIME
/******************************************************************
 * Here the current time is loaded and the time difference 
 * to beginning of the cycle is calculated and printed.
 * ****************************************************************/
        print_time(&totalStartFPS, "Total time: ");
#endif
        
#if defined(LOGCONSOLE) || defined(LOGOBJECTS) || defined(CLOCK_RUNTIME)
        std::cout << std::endl;
#endif
        
#if defined(SHOW_IM) || defined(SHOW_DEPTH)
/******************************************************************************************************
 * cv::waitKey is needed by cv::imshow.
 * In this case it listens for 1ms if a key is pressed while a window created by cv::imshow is focused.
 * If the key was ESC, the infinite while loop ends.
 * ****************************************************************************************************/
        int keycode = cv::waitKey(1) & 0xff;
        if(keycode == 27)
        {
            break;
        }
#endif
        
    }

/********************************************************
 * Cleaning up used resources and exiting the application
 * ******************************************************/
#ifdef USESHM
    #ifdef GPIO_RUNTIME            
        retval = cleanup(imgBufferRGB, imgBufferRGBAf, &p, semptr, coordinates, shmid, 1);
    #else
        retval = cleanup(imgBufferRGB, imgBufferRGBAf, &p, semptr, coordinates, shmid);
    #endif  
#else
    #ifdef GPIO_RUNTIME            
        retval = cleanup(imgBufferRGB, imgBufferRGBAf, &p, 1);
    #else
        retval = cleanup(imgBufferRGB, imgBufferRGBAf, &p);
    #endif  
#endif
    
#ifdef LOGCONSOLE
    std::cout << "Exiting now..." << std::endl;
#endif

    return retval;
}
