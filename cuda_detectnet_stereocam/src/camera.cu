#include "hip/hip_runtime.h"
#include "camera.h"

/**********************************************************************************************
 * This function initializes the camera.
 * It enables the color and depth streams with the defined resolution and framerate.
 * The resolution and framerate are defined through #RESOLUTION in "global_defines.h".
 * Then it starts the camera by starting the rs2::pipeline with the given rs2::config.
 * 
 * Input parameters:
 *  - rs2::config *cfg: Configuration for the rs2::pipeline.
 *  - rs2::pipeline *p: rs2::pipeline for starting the camera with the given rs2::config.
 *  - rs2::pipeline_profile *prof: Profile that holds the started pipeline after this function.
 * 
 * Return values:
 *  - rs2::pipeline_profile *prof: Profile that holds the started pipeline.
 *          The profile is needed for setting the maximum laser power.
 * ********************************************************************************************/
void camera_init(rs2::config *cfg, rs2::pipeline *p, rs2::pipeline_profile *prof)
{
    cfg->enable_stream(RS2_STREAM_DEPTH, WIDTH, HEIGHT, RS2_FORMAT_Z16, FRAMERATE);
    
    // Color format is set to BGR as OpenCV is used for the visualization it OpenCV uses BGR
    cfg->enable_stream(RS2_STREAM_COLOR, WIDTH, HEIGHT, RS2_FORMAT_BGR8, FRAMERATE);
    
    *prof = p->start(*cfg);
}

/***************************************************************************************************************
 * This function obtains the camera intrinsics of the color frame.
 * The intrinsics are needed for calculating the 3D coordinates of a pixel in the color frame for a given depth.
 * 
 * Input parameters:
 *  - rs2::frameset *frames: The rs2::frameset that holds the frames after grabbing them from the pipeline.
 *  - rs2::pipeline *p: The rs2::pipeline that interfaces with the camera.
 * 
 * Return values:
 *  - const rs2_intrinsics color_intrin: These are the intrinsic parameters of the color camera.
 * *************************************************************************************************************/
rs2_intrinsics get_camera_intrinsics(rs2::frameset *frames, rs2::pipeline *p)
{
    rs2::stream_profile cprof;
    
    *frames = p->wait_for_frames();
    rs2::video_frame color = frames->get_color_frame();
    (void)frames->get_depth_frame();
    
    cprof = color.get_profile();
    rs2::video_stream_profile cvsprof(cprof);
    const rs2_intrinsics color_intrin = cvsprof.get_intrinsics();
    
    return color_intrin;
}

/******************************************************************************************************************
 * This function grabs the color and depth frames from the camera.
 * For calculating the 3D coordinates for detected objects in the color frame, 
 * it is necessary to align the depth frame to the color frame.
 * This ensures that both frames have the same coordinate system 
 * and the pixels from the color frame can be used for getting the depth in the depth frame for the same position.
 * After grabbing the frames, the color frame is converted to an OpenCV Mat for object detection and visualization.
 * 
 * Input parameters:
 *  - rs2::frameset *frames: The rs2::frameset that holds the frames after grabbing them from the pipeline.
 *  - rs2::pipeline *p: The rs2::pipeline that interfaces with the camera.
 *  - rs2::align *align_to_color: rs2::align object that aligns the color and depth frame.
 *  - cv::Mat *image: OpenCV Mat from the calling function to which the color frame should be copied.
 * 
 * Return values:
 *  - rs2::depth_frame depth: This is the depth frame.
 *  - cv::Mat *image: OpenCV Mat that holds the color image.
 * ****************************************************************************************************************/
rs2::depth_frame get_frames(rs2::frameset *frames, rs2::pipeline *p, rs2::align *align_to_color, cv::Mat *image)
{
    float w, h;
    
/***************************************************
 * Grabbing and aligning the frames from the camera.
 * *************************************************/
    *frames = p->wait_for_frames();
#ifndef FILTER
/****************************************************************************************
 * In "https://github.com/IntelRealSense/librealsense/issues/1207#issuecomment-367718085" 
 * Intel recommends to align the depth frame after filtering to reduce aliasing.
 * Therefore it is done after filtering if filters are used.
 * **************************************************************************************/
    *frames = align_to_color->process(*frames);
#endif
    rs2::video_frame color = frames->get_color_frame();
    rs2::depth_frame depth = frames->get_depth_frame(); 
    
/*************************************************************************************
 * Converting the color frame to an OpenCV Mat for object detection and visualization.
 * ***********************************************************************************/
    w = color.get_width();
    h = color.get_height();
    *image = cv::Mat(cv::Size(w, h), CV_8UC3, (void*)color.get_data(), cv::Mat::AUTO_STEP);

/****************************
 * Returning the depth frame.
 * **************************/
    return depth;
}

/**********************************************************************************
 * This function sets the power of the infrared laser to its maximum value.
 * The default value in librealsense is half the power (180mW).
 * Setting the power to the maximum value increases the power consumption,
 * but it also increases the quality of the depth map.
 * For setting the maximum power, first the rs2::device 
 * and rs2::sensor are obtained from the current pipeline profile.
 * Next it is checked, if the sensor supports the option to change the laser power.
 * If it supports it, the min and max values are obtained and the max value is set.
 * 
 * Input parameters:
 *  - rs2::pipeline_profile *prof: Profile that holds the started pipeline.
 * 
 * Return values:
 *  - None
 * ********************************************************************************/
void set_max_laser_pwr(rs2::pipeline_profile *prof)
{    
/****************************************************
 * Getting the device and depth sensor of the camera.
 * **************************************************/
    rs2::device selected_device = prof->get_device();
    rs2::sensor depth_sensor = selected_device.first<rs2::depth_sensor>();

/*********************************************************
 * If the sensor supports changing the laser power, 
 * the min/max range is obtained and the max value is set.
 * Else, nothing will be changed.
 * *******************************************************/
    if (depth_sensor.supports(RS2_OPTION_LASER_POWER))
    {
        // Getting min/max range
        rs2::option_range range = depth_sensor.get_option_range(RS2_OPTION_LASER_POWER);
        depth_sensor.set_option(RS2_OPTION_LASER_POWER, range.max); // Set max power
    }
    else
    {
        std::cout << "The sensor does not support the option to change the laser power..." << std::endl;
    }
}

#ifdef FILTER
/***************************************************************************
 * This function initializes the filters with the defined values.
 * The values are defined in "global_defines.h".
 * 
 * Input parameters:
 *  - rs2::decimation_filter *dec_filter: A pointer to the decimation filter
 *  - rs2::threshold_filter *thr_filter: A pointer to the threshold filter
 *  - rs2::spatial_filter *spat_filter: A pointer to the spatial filter
 *  - rs2::temporal_filter *temp_filter: A pointer to the temporal filter
 * 
 * Return values:
 *  - Initialized filter objects.
 * *************************************************************************/
void filters_init(rs2::decimation_filter *dec_filter, rs2::threshold_filter *thr_filter, rs2::spatial_filter *spat_filter, rs2::temporal_filter *temp_filter)
{
    dec_filter->set_option(RS2_OPTION_FILTER_MAGNITUDE, DEC_MAGN);        
    thr_filter->set_option(RS2_OPTION_MIN_DISTANCE, THR_MINDIST);
    thr_filter->set_option(RS2_OPTION_MAX_DISTANCE, THR_MAXDIST);    
    spat_filter->set_option(RS2_OPTION_FILTER_MAGNITUDE, SPAT_MAGN);
    spat_filter->set_option(RS2_OPTION_FILTER_SMOOTH_ALPHA, SPAT_ALPH);
    spat_filter->set_option(RS2_OPTION_FILTER_SMOOTH_DELTA, SPAT_DELT);    
    temp_filter->set_option(RS2_OPTION_FILTER_SMOOTH_ALPHA, TEMP_ALPH);
    temp_filter->set_option(RS2_OPTION_FILTER_SMOOTH_DELTA, TEMP_DELT);
}

/***************************************************************************************************************
 * This function applies post-processing filters on the given depth frame.
 * In "https://dev.intelrealsense.com/docs/depth-post-processing" 
 * Intel recommends to apply the spatial filter and temporal filter in disparity space.
 * Therefore two rs2::disparity_transform objects are used.
 * One for transforming the frame to disparity space.
 * And the other for transforming it back to depth space after spatial and temporal filtering.
 * In "https://github.com/IntelRealSense/librealsense/issues/1207#issuecomment-367718085" 
 * Intel recommends to align the depth frame after filtering to reduce aliasing.
 * 
 * Input parameters:
 *  - rs2::depth_frame *depth: The depth frame that should be filtered.
 *  - rs2::decimation_filter *dec_filter: Decimation filter
 *  - rs2::threshold_filter *thr_filter: Threshold filter
 *  - rs2::disparity_transform *depth_to_disparity: Object to transform the frame from depth to disparity space.
 *  - rs2::spatial_filter *spat_filter: Spatial filter
 *  - rs2::temporal_filter *temp_filter: Temporal Filter
 *  - rs2::disparity_transform *disparity_to_depth: Object to transform the frame from disparity to depth space.
 *  - rs2::align *align_to_color: Object to align the filtered depth frame again to the color frame.
 * 
 * Return values:
 *  - rs2::depth_frame filtered: This is the filtered depth frame.
 * *************************************************************************************************************/
rs2::depth_frame filter_depth_frame(rs2::depth_frame *depth, rs2::decimation_filter *dec_filter, rs2::threshold_filter *thr_filter, rs2::disparity_transform *depth_to_disparity, rs2::spatial_filter *spat_filter, rs2::temporal_filter *temp_filter, rs2::disparity_transform *disparity_to_depth, rs2::align *align_to_color)
{
    rs2::depth_frame filtered = *depth;
    filtered = dec_filter->process(filtered);
    filtered = thr_filter->process(filtered);
    filtered = depth_to_disparity->process(filtered);
    filtered = spat_filter->process(filtered);
    filtered = temp_filter->process(filtered);
    filtered = disparity_to_depth->process(filtered); 
    filtered = align_to_color->process(filtered);
    
    return filtered;
}
#endif

/***************************************************************************************************************
 * This function visualizes the given depth frame.
 * First, it colors it with the given rs2::colorizer (i.e. to Jet colormap).
 * Next, it converts the rs2::depth_frame to an OpenCV Mat.
 * Finally, it visualizes the image on the screen. 
 * 
 * Input parameters:
 *  - rs2::colorizer *color_map: rs2::colorizer object that is used for applying a colorscheme to the depth map.
 *  - rs2::depth_frame *depth: The depth map that should be visualized.
 *  - std::string text: Text for naming the window.
 * 
 * Return values:
 *  - None
 * *************************************************************************************************************/
#ifdef SHOW_DEPTH
void show_depth_frame(rs2::colorizer *color_map, rs2::depth_frame *depth, std::string text)
{
    float w, h;
    
/*****************************
 * Colorizing the depth frame.
 * ***************************/
    rs2::frame depthframe = color_map->process(*depth);
 
/***************************************************
 * Converting the rs2::depth_frame to an OpenCV Mat.
 * *************************************************/
    w = depthframe.as<rs2::video_frame>().get_width();
    h = depthframe.as<rs2::video_frame>().get_height();
    cv::Mat depth_img = cv::Mat(cv::Size(w, h), CV_8UC3, (void*)depthframe.get_data(), cv::Mat::AUTO_STEP);
    
#ifdef COPTER
/*****************************************************************************************
 * On the copter, the camera is rotated by 180° and so the image also needs to be rotated.
 * This is essential for the visualization, but also for the object detection.
 * ***************************************************************************************/
#ifdef LOGCONSOLE
    std::cout << "Rotating depth image for the copter..." << std::endl;
#endif
    rotate(depth_img, depth_img, 1);
#endif

/*************************************
 * Visualizing the created OpenCV Mat.
 * ***********************************/
    cv::imshow(text + " Depth Map", depth_img);
}
#endif
