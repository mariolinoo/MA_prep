#include "hip/hip_runtime.h"
#include "utils.h"

/******************************************************
 * This is a signal handler that catches SIGINT signals
 * and tells the application to exit.
 * ****************************************************/
void sig_handler(int sig)
{
    flag = 1;
}

#ifdef REALSENSE
/***********************************************************
 * This function cleans up the used and allocated resources.
 * *********************************************************/
    #if defined(USESHM) && defined(GPIO_RUNTIME)
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, rs2::pipeline *p, sem_t *semptr, XYZ *coordinates, int shmid, int gpio)
    #elif defined(USESHM) && !defined(GPIO_RUNTIME)
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, rs2::pipeline *p, sem_t *semptr, XYZ *coordinates, int shmid)
    #elif defined(GPIO_RUNTIME) && !defined(USESHM)
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, rs2::pipeline *p, int gpio)
    #else
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, rs2::pipeline *p)
    #endif
#else
    #if defined(USESHM) && defined(GPIO_RUNTIME)
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, TaraXLSDK::TaraXLCamList *taraxlCamList, sem_t *semptr, XYZ *coordinates, int shmid, int gpio)
    #elif defined(USESHM) && !defined(GPIO_RUNTIME)
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, TaraXLSDK::TaraXLCamList *taraxlCamList, sem_t *semptr, XYZ *coordinates, int shmid)
    #elif defined(GPIO_RUNTIME) && !defined(USESHM)
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, TaraXLSDK::TaraXLCamList *taraxlCamList, int gpio)
    #else
    int cleanup(uchar3* imgBufferRGB, float4* imgBufferRGBAf, TaraXLSDK::TaraXLCamList *taraxlCamList)
    #endif
#endif
{
    int retval = EXIT_SUCCESS;
/**************************************************
 * The allocated memory on the GPU is released here
 * ************************************************/
#ifdef LOGCONSOLE
    std::cout << "Releasing allocated GPU memory..." << std::endl;
#endif
    if(imgBufferRGB != NULL)
    {
        if(hipFree(imgBufferRGB) == hipErrorInvalidValue)
        {
            std::cerr << "Error freeing imgBufferRGB..." << std::endl;
            retval = EXIT_FAILURE;
        }
    }
    if(imgBufferRGBAf != NULL)
    {
        if(hipFree(imgBufferRGBAf) == hipErrorInvalidValue)
        {
            std::cerr << "Error freeing imgBufferRGBAf..." << std::endl;
            retval = EXIT_FAILURE;
        }
    }
       
#ifdef LOGCONSOLE
    std::cout << "Stopping camera..." << std::endl;
#endif
/********************************
 * This command stops the camera.
 * ******************************/
 #ifdef REALSENSE
    try
    {
        p->stop();
    }
    catch(const rs2::error e)
    {
        // camera was not started yet, no action needed in cleanup
    }
#else
    for(int i = 0 ; i < taraxlCamList->size() ; i++)
    {
        taraxlCamList->at(i).disconnect();
    }
#endif

/****************************************************************************
 * The semaphore is closed and unlinked
 * (removed from the kernel once all applications that use it are shut down).
 * Also, the shared memory is detached and removed.
 * **************************************************************************/
#ifdef USESHM
    #ifdef LOGCONSOLE
    std::cout << "Closing semaphore and detaching shared memory..." << std::endl;
    #endif
    if(semptr != NULL)
    {
        if(sem_close(semptr) == -1)
        {
            perror("sem_close");
            retval = EXIT_FAILURE;
        }
    #ifdef REALSENSE
        if(sem_unlink("sem_rs") == -1)
    #else
         if(sem_unlink("sem_sc") == -1)   
    #endif
        {
            perror("sem_unlink");
            retval = EXIT_FAILURE;
        }
    }
    if(coordinates != NULL)
    {
        if(shmdt(coordinates) == -1)
        {
            perror("shmdt");
            retval = EXIT_FAILURE;
        }
    }
    if(shmid != -1)
    {
        if(shmctl(shmid, IPC_RMID, NULL) == -1)
        {
            perror("shmctl");
            retval = EXIT_FAILURE;
        }
    }
#endif
    
#ifdef GPIO_RUNTIME
    #ifdef LOGCONSOLE
    std::cout << "Releasing GPIOs..." << std::endl;
    #endif
    
/***************************************************************************
 * Here the GPIOs are unexported which releases them for other applications.
 * In case of an error, the error message is printed in the subfunctions.
 * *************************************************************************/
    if(gpio == 1)
    {
        if(gpio_release() != 0)
        {
            retval = EXIT_FAILURE;
        }
    }
#endif

    return retval;
}
